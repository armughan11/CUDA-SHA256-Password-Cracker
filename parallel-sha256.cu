#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include "sha256.cuh"

#define HASH_LENGTH 32 // Define the hash length
#define MAX_PASSWORD_LENGTH 10 // Define the maximum password length
#define THREADS_PER_BLOCK 256 // Number of threads per block
#define NUMBER_OF_BLOCKS 512 // Number of blocks

__constant__ char alphabet[63]; // The array of possible characters in a password

// Password generator function
__device__ void password_generator(char *password, unsigned long long index, int password_length) {
    // Generate a password from the given index
    for (int i = 0; i < password_length; i++) {
        password[i] = alphabet[index % 63];
        index /= 63;
    }
    password[password_length] = '\0';
}

// Memory comparison function
__device__ int device_memcmp(const unsigned char* a, const unsigned char* b, int size) {
    // Compare two given arrays of the specified size
    for (int i = 0; i < size; i++) {
        if (a[i] != b[i]) {
            return 1;
        }
    }
    return 0;
}

__device__ int found_flag = 0; // Flag to check if password has been found

__global__ void password_cracker(char *device_password, int original_password_length, int generated_password_length, unsigned long long start_index, unsigned long long password_count, uint32_t *processedPasswords) {
    // Early return if password has been found
    if (found_flag) {
        return;
    }
    unsigned long long thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    // Only continue if the current thread index is within the total number of passwords
    if (thread_index < password_count) {
        char password[MAX_PASSWORD_LENGTH + 1];
        // Generate a password using the current thread index
        password_generator(password, start_index + thread_index, generated_password_length);

        unsigned char hash[HASH_LENGTH];
        unsigned char device_hash[HASH_LENGTH];
        SHA256_CTX sha256;

        // Calculate hash of the generated password
        sha256_init(&sha256);
        sha256_update(&sha256, (BYTE*)password, generated_password_length); // Use generated_password_length instead of MAX_PASSWORD_LENGTH
        sha256_final(&sha256, hash);

        // Calculate hash of the original password
        sha256_init(&sha256);
        sha256_update(&sha256, (BYTE*)device_password, original_password_length);
        sha256_final(&sha256, device_hash);

        // Compare the two hashes
        if (device_memcmp(hash, device_hash, HASH_LENGTH) == 0) {
            // If hashes match, password has been found
            printf("Password found: %s\n", password);
            found_flag = 1;
            return;
        }
    }
    // Increment the processed passwords counter
    atomicAdd(processedPasswords, 1);
}

// Function to get the current time in milliseconds
long long get_time_in_seconds() {
    struct timeval time;
    gettimeofday(&time, NULL);
    return time.tv_sec * 1000LL + time.tv_usec / 1000;
}

// Main function
int main(int argc, char** argv) {
    // Set the array of possible characters
    char host_alphabet[63] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
    // Copy the array to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(alphabet), host_alphabet, sizeof(host_alphabet));

    char host_password[MAX_PASSWORD_LENGTH+1] = "armug"; // The password to crack

     if(argc > 1) {
        strncpy(host_password, argv[1], MAX_PASSWORD_LENGTH); // If a password argument is provided, use it. strncpy prevents buffer overflow.
        host_password[MAX_PASSWORD_LENGTH] = '\0'; // Ensures null termination.
    }
    // Device memory allocation and copying for the original password
    char* device_password;
    int password_length = strlen(host_password) + 1; // +1 for the null terminator
    hipMalloc((void**)&device_password, password_length * sizeof(char)); // Allocate memory for the full password
    hipMemcpy(device_password, host_password, password_length * sizeof(char), hipMemcpyHostToDevice); // Copy the full password

    // Calculate the total number of possible passwords
    unsigned long long total_passwords = powl(strlen(host_alphabet), MAX_PASSWORD_LENGTH);

    // Get the start time
    long long start_time = get_time_in_seconds();

    // Device memory allocation and copying for the counter of processed passwords
    uint32_t *device_processedPasswords, host_processedPasswords = 0;
    hipMalloc((void**)&device_processedPasswords, sizeof(uint32_t));
    hipMemcpy(device_processedPasswords, &host_processedPasswords, sizeof(uint32_t), hipMemcpyHostToDevice);

    // Loop over all possible password lengths
    for (int password_length = 1; password_length <= MAX_PASSWORD_LENGTH; password_length++) {
        // Calculate the total number of passwords of the current length
        unsigned long long password_count = powl(strlen(host_alphabet), password_length);

        // Loop over all possible starting indices for the password generation
        for (unsigned long long start_index = 0; start_index < password_count; start_index += THREADS_PER_BLOCK * NUMBER_OF_BLOCKS) {
            // Calculate the number of remaining passwords and the number of blocks
            unsigned long long remaining_passwords = min(password_count - start_index, (unsigned long long)(THREADS_PER_BLOCK * NUMBER_OF_BLOCKS));
            unsigned long long block_count = min((remaining_passwords + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (unsigned long long)NUMBER_OF_BLOCKS);
            
            // Call the password cracker kernel
            password_cracker<<<block_count, THREADS_PER_BLOCK>>>(device_password, strlen(host_password), password_length, start_index, remaining_passwords, device_processedPasswords);
            
            hipDeviceSynchronize(); // Synchronize device
            
            // Check if password has been found
            int host_found_flag;
            hipMemcpyFromSymbol(&host_found_flag, HIP_SYMBOL(found_flag), sizeof(int), 0, hipMemcpyDeviceToHost);
            if (host_found_flag) {
                //printf("Password found!\n");
                long long elapsed = get_time_in_seconds() - start_time;
                //printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r", host_processedPasswords, ((float) elapsed) / 1000.0, (unsigned long) ((double) host_processedPasswords / (double) elapsed) * 1000);
                break;
            }
            // Copy the number of processed passwords from device to host
            hipMemcpy(&host_processedPasswords, device_processedPasswords, sizeof(uint32_t), hipMemcpyDeviceToHost);
            // Print progress information
            long long elapsed = get_time_in_seconds() - start_time;
            printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r\n", host_processedPasswords, ((float) elapsed) / 1000.0, (unsigned long) ((double) host_processedPasswords / (double) elapsed) * 1000);
            fflush(stdout); // Flush the output buffer to ensure immediate printing
        }
        if (found_flag) {
            break;
        }
    }

    printf("\n");
    long long elapsed = get_time_in_seconds() - start_time;

    printf("Hashes processed (%'lu) Time Taken in seconds (%'f) Avg Hashes/sec (%'lu)\n", host_processedPasswords, ((float) elapsed) / 1000.0, (unsigned long) ((double) host_processedPasswords / (double) elapsed) * 1000);
    // Free the device memory
    hipFree(device_password);
    hipFree(device_processedPasswords);

    return 0;
}
